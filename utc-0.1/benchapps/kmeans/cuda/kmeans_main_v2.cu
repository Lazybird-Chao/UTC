#include "hip/hip_runtime.h"
/*
 * kmeans_mian_v2.cu
 *
 *  Created on: Jan 19, 2017
 *      Author: chao
 */
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <assert.h>

#include "../../common/helper_getopt.h"
#include "../../common/helper_timer.h"
#include "../../common/helper_err.h"

#include "file_io.h"
#include "kmeans_kernel_v2.h"

#define FTYPE float
#define PREC 300 // max iteration times


/*
*	Function: usage
*	---------------
*	Prints information on how to call the program.
*/
static void usage(char *argv0) {
    char *help =
        "Usage: %s [switches] -i filename -n num_clusters [OPTIONS]\n"
        "       -i filename    : file containing data to be clustered\n"
        "       -b             : input file is in binary format (default no)\n"
        "       -n num_clusters: number of clusters (K must be > 1)\n"
        "       -o filename    : write output to file\n";
    fprintf(stderr, help, argv0);
    exit(-1);
}

template<typename T>
T** create_array_2d(int height, int width) {
	T** ptr;
	int i;
	ptr = (T**)calloc(height, sizeof(T*));
	assert(ptr != NULL);
	ptr[0] = (T*)calloc(width * height, sizeof(T));
	assert(ptr[0] != NULL);
	/* Assign pointers correctly */
	for(i = 1; i < height; i++)
		ptr[i] = ptr[i-1] + width;
	return ptr;
}


/*---< main() >-------------------------------------------------------------*/
int main(int argc, char **argv) {

	int numClusters, numCoords, numObjs;

    int     isBinaryFile;
    int    *membership;    /* [numObjs] */
    char   *filename, *outfile;
    FTYPE **objects;       /* [numObjs][numCoords] data objects */
    FTYPE **clusters;      /* [numClusters][numCoords] cluster center */
    FTYPE   threshold;
    double  io_timing, clustering_timing;

    /* some default values */
    numClusters       = 1;		/* Amount of cluster centers */
    threshold         = 0.001; 	/* Percentage of objects that need to change membership for the clusting to continue */
    isBinaryFile      = 0;		/* 0 if the input file is in ASCII format, 1 for binary format */
    filename          = NULL;	/* Name of the input file */
    outfile           = NULL;

	/* Parse command line options */
    int     opt;
	extern char   *optarg;
	extern int     optind;
    while ( (opt=getopt(argc,argv,"o:i:n:b"))!= EOF) {
        switch (opt) {
            case 'i': filename=optarg;
                      break;
            case 'b': isBinaryFile = 1;
                      break;
            case 'n': numClusters = atoi(optarg);
                      break;
            case 'h': usage(argv[0]);
                      break;
            case 'o': outfile = optarg;
                      break;
            default: usage(argv[0]);
                      break;
        }
    }

    if (filename == NULL) usage(argv[0]);


    double t1, t2;
    std::cout<<"Reading objecs file"<<std::endl;
    t1 = getTime();
    /* Read input data points from given input file */
    objects = file_read<FTYPE>(isBinaryFile, filename, &numObjs, &numCoords);
    assert(objects != NULL);
	t2 = getTime();
	io_timing        	= t2 - t1;


    membership = (int*) malloc(numObjs * sizeof(int));
    int *new_membership = (int*) malloc(numObjs * sizeof(int));
    clusters = create_array_2d<FTYPE>(numClusters, numCoords);
    FTYPE **new_clusters = create_array_2d<FTYPE>(numClusters, numCoords);
    int *new_clustersize = (int*)calloc(numClusters, sizeof(int));
    /* Pick first numClusters elements of objects[] as initial cluster centers */
	for (int i=0; i < numClusters; i++)
		for (int j=0; j < numCoords; j++)
			clusters[i][j] = objects[i][j];

	/* Initialize membership, no object belongs to any cluster yet */
	for (int i = 0; i < numObjs; i++)
		membership[i] = -1;

    /*
     * create gpu memory
     */
	hipSetDevice(0);
    FTYPE *objects_d;
    int *membership_d;
    FTYPE *clusters_d;
    checkCudaErr(hipMalloc(&objects_d, sizeof(FTYPE)*numObjs*numCoords));
    checkCudaErr(hipMalloc(&membership_d, sizeof(int)*numObjs));
    checkCudaErr(hipMalloc(&clusters_d, sizeof(FTYPE)*numClusters*numCoords));

    /*
     * copy data in
     */
    t1 = getTime();
    checkCudaErr(hipMemcpy(objects_d, objects[0], sizeof(FTYPE)*numObjs*numCoords, hipMemcpyHostToDevice));
    t2 = getTime();
    double copyinTime = t2- t1;

    /*
	 * copy in new clusters data
	 */
	t1 = getTime();
	checkCudaErr(hipMemcpy(clusters_d, clusters[0], sizeof(FTYPE)*numClusters*numCoords, hipMemcpyHostToDevice));
	t2 = getTime();
	copyinTime += t2-t1;

    /*
     * kernel computing
     */
	std::cout<<"Start clustering..."<<std::endl;
	double kernelTime =0;
	double copyoutTime = 0;
	double hostCompTime = 0;

	int batchPerThread = 16;
	int blocksize = 256;
	int gridsize = (numObjs + blocksize*batchPerThread -1)/(blocksize*batchPerThread);
	dim3 membership_kernel_block(blocksize, 1, 1);
	dim3 membership_kernel_grid(gridsize, 1, 1);
	FTYPE *new_clusters_reduce_d;
	int *new_clusters_size_reduce_d;
	int *new_clusters_size_d;
	int *change_count_reduce_d;
	int *change_count_reduce;
	checkCudaErr(hipMalloc(&new_clusters_reduce_d, sizeof(FTYPE)*numClusters*numCoords*gridsize));
	checkCudaErr(hipMalloc(&new_clusters_size_reduce_d, sizeof(int)*numClusters*gridsize));
	checkCudaErr(hipMalloc(&new_clusters_size_d, sizeof(int)*numClusters));
	checkCudaErr(hipMalloc(&change_count_reduce_d, sizeof(int)*gridsize));
	change_count_reduce = (int*)malloc(sizeof(int)*gridsize);

	dim3 new_clusters_size_kernel_block(blocksize, 1, 1);
	dim3 new_clusters_size_kernel_grid((numClusters+ blocksize-1)/blocksize, 1, 1);

	dim3 new_clusters_kernel_block(blocksize, 1, 1);
	int gridsize2 = (numClusters*numCoords + blocksize-1)/(blocksize);
	dim3 new_clusters_kernel_grid(gridsize2, 1, 1);


	int changedObjs =0;
	int loopcounters = 0;
	do{

		std::cout<<__LINE__<<std::endl;
		t1 = getTime();
		membership_kernel<<<membership_kernel_grid,
				membership_kernel_block>>>(objects_d, numCoords, numObjs, numClusters,
                          clusters_d, membership_d, batchPerThread,
                          new_clusters_reduce_d, new_clusters_size_reduce_d,
                          change_count_reduce_d);
		hipDeviceSynchronize();
		std::cout<<__LINE__<<std::endl;
		new_clusters_size_kernel<<<new_clusters_size_kernel_grid,
					new_clusters_size_kernel_block>>>(new_clusters_size_reduce_d,
							new_clusters_size_d,
							numClusters, gridsize);
		hipDeviceSynchronize();
		new_clusters_kernel<<<new_clusters_kernel_grid,
				new_clusters_kernel_block>>>(new_clusters_reduce_d,
						clusters_d,
						new_clusters_size_d,
						numClusters, numCoords, gridsize);
		hipDeviceSynchronize();
		checkCudaErr(hipGetLastError());
		t2 = getTime();
		kernelTime += t2-t1;
		std::cout<<__LINE__<<std::endl;
		/*
		 * copy out change obj counts
		 */
		t1 = getTime();
		checkCudaErr(hipMemcpy(change_count_reduce, change_count_reduce_d, sizeof(int)*gridsize, hipMemcpyDeviceToHost));
		t2 = getTime();
		copyoutTime += t2-t1;

		/*
		 * compute new clusters
		 */
		t1 = getTime();
		changedObjs = 0;
		for(int i=0; i<gridsize; i++)
			changedObjs += change_count_reduce[i];
		t2 = getTime();
		hostCompTime += t2-t1;

	}while(loopcounters++ < PREC && (FTYPE)changedObjs/numObjs > threshold );


    /* Memory cleanup */
    free(objects[0]);
	free(objects);
    free(membership);
    free(new_clusters[0]);
    free(new_clusters);
    free(new_membership);
    free(new_clustersize);
    hipFree(objects_d);
    hipFree(membership_d);
    hipFree(clusters_d);

    hipFree(new_clusters_reduce_d);
    hipFree(new_clusters_size_reduce_d);
    hipFree(new_clusters_size_d);
    hipFree(change_count_reduce_d);
    free(change_count_reduce);


    t1 = getTime();
    if(outfile != NULL) {
        int l;
        FILE* fp = fopen(outfile, "w");
        for(int j = 0; j < numClusters; j++) {
            fprintf(fp, "Cluster %d: ", j);
            for(l = 0; l < numCoords; l++)
                fprintf(fp, "%f ", clusters[j][l]);
            fprintf(fp, "\n");
        }
        fclose(fp);
    }
    t2 = getTime();
    io_timing += t2 - t1;
    free(clusters[0]);
    free(clusters);

    /* Print performance numbers on stdout */

    printf("\n---- kMeans Clustering ----\n");
    printf("Input file:     %s\n", filename);
    printf("numObjs       = %d\n", numObjs);
    printf("numCoords     = %d\n", numCoords);
    printf("numClusters   = %d\n", numClusters);
    printf("threshold     = %.4f\n", threshold);

    printf("Iterations     	   = %d\n", loopcounters);
    printf("I/O time           = %10.4f sec\n", io_timing);
    printf("copyin time        = %10.4f sec\n", copyinTime);
    printf("copyout time       = %10.4f sec\n", copyoutTime);
    printf("gpu kernel time    = %10.4f sec\n", kernelTime);
    printf("host compute time  = %10.4f sec\n", hostCompTime);
    clustering_timing = copyinTime + copyoutTime + kernelTime + hostCompTime;
    printf("Computation timing = %10.4f sec\n", clustering_timing);

    return(0);
}



