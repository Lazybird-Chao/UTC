/*
 * c-ray_task_sgpu.cu
 *
 *  Created on: Mar 24, 2017
 *      Author: chao
 */
#include "c-ray_task_mgpu.h"
#include "c-ray_kernel_v2.h"
#include "../../../common/helper_err.h"

__device__  global_vars g_vars_d;
__device__  vec3_t lights_d[MAX_LIGHTS];
__device__  vec2_t urand_d[NRAN];
__device__  int irand_d[NRAN];

thread_local int crayMGPU::local_yres;
thread_local int crayMGPU::local_startYresIndex;

void crayMGPU::initImpl(global_vars g_vars,
		sphere_array_t obj_array,
		uint32_t *pixels,
		vec3_t *lights){
	if(__localThreadId ==0){
		std::cout<<"task: "<<getCurrentTask()->getName()<<" begin init ...\n";

		this->g_vars = g_vars;
		this->obj_array = obj_array;
		this->pixels = pixels;
		this->lights = lights;
	}
	intra_Barrier();
	int yresPerThread = g_vars.yres / __numLocalThreads;
	if(__localThreadId < g_vars.yres % __numLocalThreads){
		local_yres = yresPerThread +1;
		local_startYresIndex = __localThreadId *(yresPerThread+1);
	}
	else{
		local_yres = yresPerThread;
		local_startYresIndex = __localThreadId*yresPerThread + g_vars.yres % __numLocalThreads;
	}

	intra_Barrier();
	if(__localThreadId ==0){
		std::cout<<"task: "<<getCurrentTask()->getName()<<" finish initImpl.\n";
	}
}

void crayMGPU::runImpl(double runtime[][4], MemType memtype){
	if(__localThreadId == 0){
		std::cout<<getCurrentTask()->getName()<<" begin run ..."<<std::endl;
	}
	Timer timer, timer0;
	double totaltime;

	int xres = g_vars.xres;	//column
	int yres = g_vars.yres;	//row
	GpuData<unsigned int> partial_pixels_d(xres*local_yres);
	GpuData<vec3_t> obj_array_pos(g_vars.obj_count);
	GpuData<material_t> obj_array_mat(g_vars.obj_count);
	GpuData<FTYPE> obj_array_rad(g_vars.obj_count);
	obj_array_pos.initH(obj_array.pos);
	obj_array_mat.initH(obj_array.mat);
	obj_array_rad.initH(obj_array.rad);

	vec2_t urand[NRAN];
	int irand[NRAN];
	for(int i=0; i<NRAN; i++) urand[i].x = (double)rand() / RAND_MAX - 0.5;
	for(int i=0; i<NRAN; i++) urand[i].y = (double)rand() / RAND_MAX - 0.5;
	for(int i=0; i<NRAN; i++) irand[i] = (int)(NRAN * ((double)rand() / RAND_MAX));

	/*
	 * data in
	 */
	timer0.start();
	timer.start();
	obj_array_pos.sync();
	obj_array_mat.sync();
	obj_array_rad.sync();


	checkCudaErr(
			hipMemcpyToSymbol(HIP_SYMBOL(lights_d), lights, sizeof(vec3_t)*MAX_LIGHTS, 0,
					hipMemcpyHostToDevice));
	checkCudaErr(
			hipMemcpyToSymbol(HIP_SYMBOL(urand_d), urand, sizeof(vec2_t)*NRAN, 0,
					hipMemcpyHostToDevice));
	checkCudaErr(
			hipMemcpyToSymbol(HIP_SYMBOL(irand_d), irand, sizeof(int)*NRAN, 0,
					hipMemcpyHostToDevice));
	checkCudaErr(
			hipMemcpyToSymbol(HIP_SYMBOL(g_vars_d), (void*)&g_vars, sizeof(g_vars), 0,
					hipMemcpyHostToDevice));
	double copyinTime = timer.stop();

	/*
	 * call kernel
	 */
	size_t stacksize;
	cudaThreadGetLimit(&stacksize, hipLimitStackSize);
	stacksize = 1024*4;
	cudaThreadSetLimit(hipLimitStackSize, stacksize);
	dim3 block(16, 16, 1);
	dim3 grid((xres+block.x-1)/block.x, (local_yres+block.y-1)/block.y,1);
	timer.start();
	render_kernel<<<grid, block, 0, __streamId>>>(
			partial_pixels_d.getD(true),
			obj_array_pos.getD(),
			obj_array_mat.getD(),
			obj_array_rad.getD(),
			local_startYresIndex
			);
	checkCudaErr(hipGetLastError());
	checkCudaErr(hipStreamSynchronize(__streamId));
	double kernelTime = timer.stop();

	/*
	 *
	 */
	timer.start();
	partial_pixels_d.sync();
	double copyoutTime = timer.stop();
	totaltime = timer0.stop();
	partial_pixels_d.fetch(pixels+local_startYresIndex*xres);

	//runtime[0] = copyinTime + copyoutTime + kernelTime;
	runtime[__localThreadId][0] = totaltime;
	runtime[__localThreadId][1]= kernelTime;
	runtime[__localThreadId][2]= copyinTime;
	runtime[__localThreadId][3]= copyoutTime;

	if(__localThreadId ==0){
		std::cout<<"task: "<<getCurrentTask()->getName()<<" finish runImpl.\n";
	}
}

