#include "hip/hip_runtime.h"
/*
 * rotate_task_sgpu.cu
 *
 *  Created on: Mar 15, 2017
 *      Author: chao
 */
#include "rotate_task_mgpu.h"
#include "rotate_kernel.h"
#include "../../../common/helper_err.h"
#include <iostream>



void RotateMGPU::initImpl(Image* srcImg, Image* dstImg, int angle){
	if(__localThreadId ==0){
		std::cout<<"begin init ...\n";
		this->srcImg = srcImg;
		this->dstImg = dstImg;
		this->angle = angle;

		/*
		 * compute the out image's size
		 */
		float2 ul, ur, ll, lr;
		float xc = (float)srcImg->getWidth() / 2.0;
		float yc = (float)srcImg->getHeight() / 2.0;
		ul.x = -xc;
		ul.y = yc;
		ur.x = xc;
		ur.y = yc;
		ll.x = -xc;
		ll.y = -yc;
		lr.x = xc;
		lr.y = -yc;
		float2 outCorner[4];
		rotatePoint(ul, outCorner[0], angle);
		rotatePoint(ur, outCorner[1], angle);
		rotatePoint(ll, outCorner[2], angle);
		rotatePoint(lr, outCorner[3], angle);
		//compute the out image's size
		float maxW = outCorner[0].x;
		float minW = outCorner[0].x;
		float maxH = outCorner[0].y;
		float minH = outCorner[0].y;
		for(int i=1; i<4; i++){
			if(outCorner[i].x > maxW)
				maxW = outCorner[i].x;
			if(outCorner[i].x< minW)
				minW = outCorner[i].x;
			if(outCorner[i].y > maxH)
				maxH = outCorner[i].y;
			if(outCorner[i].y< minH)
				minH = outCorner[i].y;
		}
		int outH = (int)maxH-minH;
		int outW = (int)maxW-minW;
		dstImg->createImageFromTemplate(outW, outH, srcImg->getDepth());

	}
	intra_Barrier();
	int rows_per_thread = outH/__numLocalThreads;
	if(__localThreadId < outH % __numLocalThreads){
		num_rows = rows_per_thread+1;
		start_row = __localThreadId*(rows_per_thread+1);
		end_row = start_row + (rows_per_thread+1) -1;
	}
	else{
		num_rows = rows_per_thread;
		start_row = __localThreadId*rows_per_thread + outH % __numLocalThreads;
		end_row = start_row + rows_per_thread -1;
	}

	intra_Barrier();
	if(__localThreadId ==0){
		std::cout<<"task: "<<getCurrentTask()->getName()<<" finish initImpl.\n";
	}
}

void RotateMGPU::runImpl(double **runtime, MemType memtype){
	if(__localThreadId == 0){
		std::cout<<getCurrentTask()->getName()<<" begin run ..."<<std::endl;
	}

	Timer timer, timer0;
	double totaltime;

	GpuData<Pixel> sImg(srcImg->getWidth()*srcImg->getHeight(), memtype);
	//only create part of dImg
	GpuData<Pixel> partial_dImg(dstImg->getWidth()*num_rows, memtype);
	sImg.initH(srcImg->getPixelBuffer());
	//std::cout<<srcImg->getWidth()<<" "<<srcImg->getHeight()<<" "<<sizeof(Pixel)<<" "<<sImg.getBSize()<<std::endl;

	//std::cout<<srcImg->getWidth()<<" "<<srcImg->getHeight()<<sImg.getBSize()<<std::endl;

	/*
	 * copy data in
	 */
	timer0.start();
	timer.start();
	//memcpy(sImg.getH(true), srcImg->getPixelBuffer(), sImg.getBSize());
	sImg.syncH();
	double copyinTime = timer.stop();

	/*
	 * invoke kernel
	 */
	timer.start();
	int blocksize_x = 32;
	int blocksize_y = 16;
	int batchx = 1;
	int batchy = 1;
	dim3 block(blocksize_x, blocksize_y, 1);
	dim3 grid((dstImg->getWidth()+blocksize_x*batchx-1)/(blocksize_x*batchx),
				(num_rows+blocksize_y*batchy-1)/(blocksize_y*batchy),
				1);
	rotate_kernel<<<grid, block, 0, __streamId>>>(sImg.getD(),
									srcImg->getWidth(),
									srcImg->getHeight(),
									partial_dImg.getD(true),
									dstImg->getWidth(),
									dstImg->getHeight(),
									angle,
									start_row.load(),
									end_row.load(),
									batchx,
									batchy);
	checkCudaErr(hipGetLastError());
	//checkCudaErr(hipDeviceSynchronize());
	checkCudaErr(hipStreamSynchronize(__streamId));
	double kernelTime = timer.stop();

	/*
	 * copy data out
	 */
	timer.start();
	partial_dImg.syncD();
	//Pixel tmp = dImg.at(1000);
	//std::cout<<tmp.r<<std::endl;
	double copyoutTime = timer.stop();
	Pixel *dImg_startPtr = dstImg->getPixelBuffer() + start_row*dstImg->getWidth();
	memcpy(dImg_startPtr, partial_dImg.getH(), partial_dImg.getBSize());
	totaltime = timer0.stop();

	runtime[__localThreadId][2] = copyinTime;
	runtime[__localThreadId][3] = copyoutTime;
	runtime[__localThreadId][1] = kernelTime;
	//runtime[0] = copyinTime+copyoutTime+kernelTime;
	runtime[__localThreadId][0] = totaltime;

	intra_Barrier();
	if(__localThreadId ==0){
		std::cout<<"task: "<<getCurrentTask()->getName()<<" finish runImpl.\n";
	}

}


