/*
 * bfs_main.cu
 *
 *  Created on: Feb 28, 2017
 *      Author: chao
 *
 * The gpu version of BFS algorithm.
 * Similar to the sequential version, but use one cuda thread to process one
 * graph node in the font-wave in every iteration. So for each iteration, the
 * number of cuda threads of gpu kernel may change.
 *
 * usage:
 * 		Compile with Makefile.
 * 		run as: ./a.out -v -i inputfile -o outputfile
 * 			-v: print time info
 * 			-i: the input graph data file path
 * 			-o: output file path
 *
 *
 */


#include <cstdlib>
#include <cstdio>
#include <cstdint>
#include <climits>
#include <iostream>
#include <iomanip>
#include <vector>

#include "../../common/helper_getopt.h"
#include "../../common/helper_timer.h"
#include "../../common/helper_err.h"
#include "bfs_main.h"
#include "bfs_comm_data.h"
#include "bfs_kernel.h"


int main(int argc, char*argv[]){
	bool printTime = false;
	char* input_path = NULL;
	char* output_path=NULL;

	/*
	 * parse arguments
	 */
	int opt;
	extern char* optarg;
	extern int optind;
	while((opt=getopt(argc, argv, "vi:o:"))!=EOF){
		switch(opt){
		case 'v':
			printTime = true;
			break;
		case 'i':
			input_path = optarg;
			break;
		case 'o':
			output_path = optarg;
			break;
		case '?':
			break;
		default:
			break;
		}
	}
	if(input_path == NULL){
		std::cerr<<"Need input file path with -i !!!"<<std::endl;
		return 1;
	}

	/*
	 * read input file and initialize the graph data
	 */
	std::cout<<"Read graph data ..."<<std::endl;
	Node_t *graph_nodes;
	Edge_t *graph_edges;
	int total_graph_nodes;
	int total_graph_edges;
	int source_nodeid;
	initGraphFromFile(input_path, graph_nodes, graph_edges,
			total_graph_nodes, total_graph_edges, source_nodeid);
	//std::cout<<total_graph_nodes<<" "<<total_graph_edges<<std::endl;
	int *shortestPath = new int[total_graph_nodes];
	for(int i=0; i<total_graph_nodes; i++){
		shortestPath[i] = INT_MAX;
	}
	shortestPath[source_nodeid] = 0;

	double t1, t2;
	/*
	 * create gpu memory
	 */
	hipSetDevice(0);
	Node_t *graph_nodes_d;
	Edge_t *graph_edges_d;
	int *shortestPath_d;
	checkCudaErr(hipMalloc(&graph_nodes_d,
			total_graph_nodes*sizeof(Node_t)));
	checkCudaErr(hipMalloc(&graph_edges_d,
			total_graph_edges*sizeof(Edge_t)));
	checkCudaErr(hipMalloc(&shortestPath_d,
			total_graph_nodes*sizeof(int)));

	/*
	 * copyin data
	 */
	t1 = getTime();
	checkCudaErr(hipMemcpy(graph_nodes_d,
			graph_nodes,
			total_graph_nodes*sizeof(Node_t),
			hipMemcpyHostToDevice));
	checkCudaErr(hipMemcpy(graph_edges_d,
				graph_edges,
				total_graph_edges*sizeof(Edge_t),
				hipMemcpyHostToDevice));
	checkCudaErr(hipMemcpy(shortestPath_d,
				shortestPath,
				total_graph_nodes*sizeof(int),
				hipMemcpyHostToDevice));
	t2 = getTime();
	double copyinTime =0;
	copyinTime += t2-t1;

	/*
	 * call kernel to do bfs
	 */
	int* frontWave_d;
	int* nextWave_d;
	// allocal wave array, and assume the real wave size will not exceed
	// MAX_WAVE_SIZE during the iteration
	checkCudaErr(hipMalloc(&frontWave_d,
			MAX_WAVE_SIZE*sizeof(int)));
	checkCudaErr(hipMalloc(&nextWave_d,
			MAX_WAVE_SIZE*sizeof(int)));
	int frontWaveSize;
	int* nextWaveSize_d;
	checkCudaErr(hipMalloc((void**)&nextWaveSize_d, sizeof(int)));
	std::cout<<"start bfs processing ..."<<std::endl;

	//add source node id to frontwave to start
	t1 = getTime();
	checkCudaErr(hipMemcpy(frontWave_d, &source_nodeid,
			sizeof(int), hipMemcpyHostToDevice));
	t2= getTime();
	copyinTime += t2-t1;
	frontWaveSize = 1;

	double kernelTime=0;
	double copyoutTime=0;
	while(frontWaveSize >0){
		int reset = 0;
		t1=getTime();
		/*checkCudaErr(hipMemcpyToSymbol(HIP_SYMBOL(nextWaveSize_d), &reset, sizeof(int),
				0, hipMemcpyHostToDevice));*/
		checkCudaErr(hipMemcpy(nextWaveSize_d, &reset, sizeof(int), hipMemcpyHostToDevice));
		t2 = getTime();
		copyinTime += t2-t1;

		t1 = getTime();
		if(frontWaveSize > MAX_THREAD_PER_BLOCK){
			//std::cout<<"go multiblock ..."<<std::endl;
			dim3 block(MAX_THREAD_PER_BLOCK, 1 ,1);
			dim3 grid((frontWaveSize+MAX_THREAD_PER_BLOCK-1)/MAX_THREAD_PER_BLOCK,1,1);
			bfs_multiblocks<<<grid, block>>>(
					graph_nodes_d,
					graph_edges_d,
					shortestPath_d,
					frontWaveSize,
					frontWave_d,
					nextWave_d,
					nextWaveSize_d);
		}
		else{
			//std::cout<<"go single ..."<<std::endl;
			dim3 block(MAX_THREAD_PER_BLOCK,1,1);
			dim3 grid(1,1,1);
			bfs_singleblock<<<grid, block>>>(
					graph_nodes_d,
					graph_edges_d,
					shortestPath_d,
					frontWaveSize,
					frontWave_d,
					nextWave_d,
					nextWaveSize_d);
		}
		checkCudaErr(hipGetLastError());
		checkCudaErr(hipDeviceSynchronize());
		t2 = getTime();
		kernelTime += t2 -t1;

		t1= getTime();
		/*checkCudaErr(hipMemcpyFromSymbol(&frontWaveSize, HIP_SYMBOL(nextWaveSize_d),
				sizeof(int), 0, hipMemcpyDeviceToHost));*/
		checkCudaErr(hipMemcpy(&frontWaveSize, nextWaveSize_d, sizeof(int), hipMemcpyDeviceToHost));
		t2 = getTime();
		copyoutTime += t2-t1;
		//std::cout<<frontWaveSize<<std::endl;
		int *tmp = frontWave_d;
		frontWave_d = nextWave_d;
		nextWave_d = tmp;
	}


	/*
	 * copy result back
	 */
	t1 = getTime();
	checkCudaErr(hipMemcpy(shortestPath, shortestPath_d,
			total_graph_nodes*sizeof(int),
			hipMemcpyDeviceToHost));
	t2 = getTime();
	copyoutTime += t2-t1;
	/*
	 * write result
	 */
	if(output_path!=NULL){
		std::cout<<"write output ..."<<std::endl;
		writeOutput(output_path, shortestPath, total_graph_nodes);
	}

	delete graph_nodes;
	delete graph_edges;
	delete shortestPath;
	hipFree(graph_nodes_d);
	hipFree(graph_edges_d);
	hipFree(shortestPath_d);
	hipFree(frontWave_d);
	hipFree(nextWave_d);
	hipFree(nextWaveSize_d);

	std::cout<<"Test complete !!!"<<std::endl;
	if(printTime){
		std::cout<<"\tgraph info:"<<std::endl;
		std::cout<<"\t\tnodes: "<<total_graph_nodes<<std::endl;
		std::cout<<"\t\tedges: "<<total_graph_edges<<std::endl;
		std::cout<<"\t\tsource node id: "<<source_nodeid<<std::endl;
		std::cout<<"\ttime info: "<<std::endl;
		std::cout<<"\t\tTotal time: "<<std::fixed<<std::setprecision(4)
					<<1000*(kernelTime+copyinTime+copyoutTime)<<"(ms)"<<std::endl;
		std::cout<<"\t\tkernel time: "<<std::fixed<<std::setprecision(4)<<1000*kernelTime<<"(ms)"<<std::endl;
		std::cout<<"\t\tcopyin time: "<<std::fixed<<std::setprecision(4)<<1000*copyinTime<<"(ms)"<<std::endl;
		std::cout<<"\t\tcopyout time: "<<std::fixed<<std::setprecision(4)<<1000*copyoutTime<<"(ms)"<<std::endl;
	}
	return 0;


}

void initGraphFromFile(char* infile,
		Node_t *&nodes, Edge_t *&edges,
		int &total_nodes, int &total_edges, int &src_node){
	FILE *fp = fopen(infile, "r");
	if(!fp){
		std::cerr<<"Can't open input file !!!"<<std::endl;
		exit(1);
	}
	int x, y, count;
	count = fscanf(fp, "%d", &total_nodes);
	nodes = new Node_t[total_nodes];

	for(int i=0; i<total_nodes; i++){
		count =fscanf(fp, "%d", &x);
		count=fscanf(fp, "%d", &y);

		nodes[i].start_edgeid = x;
		nodes[i].num_edges = y;
	}

	count =fscanf(fp, "%d", &src_node);
	count =fscanf(fp, "%d", &total_edges);
	edges = new Edge_t[total_edges];
	for(int i=0; i<total_edges; i++){
		count=fscanf(fp, "%d", &x);
		count=fscanf(fp, "%d", &y);
		//edges[i].dst_nodeid = x;
		//edges[i].weight = y;
		edges[i] = x;
	}

	fclose(fp);

}

void writeOutput(
		char *outfile,
		int *spath,
		int total_nodes){
	FILE *fp = fopen(outfile, "w");
	if(!fp){
		std::cout<<"Cann't open the output file !!!"<<std::endl;
		exit(1);
	}
	fprintf(fp, "%d\n", total_nodes);
	for(int i=0; i<total_nodes; i++){
		fprintf(fp, "%d %d\n", i, spath[i]);
	}
	fclose(fp);

}


